#include "hip/hip_runtime.h"
/*This code is licensed under the:  Creative Commons - Attribution-Share Alike 2.0 UK: England & Wales  License
  Please see the full license at: http://www.bv2.co.uk/?page_id=849
  or the license certificate at:  http://creativecommons.org/licenses/by-sa/2.0/uk/
  contact Barrett@bv2.co.uk for other permissions
*/


__constant__ float d_G1D[5];


void calcGaussianCoefficients()
{
	//create the 1D kernel co-efficients
	float G1D[5];
	int g1dindex = 0;
	total=0;
	for (int z=-2;z<=2;++z)
	{
		float factor = exp(-((z*z)/(2*1.4*1.4)));   
		G1D[g1dindex] = factor;
		g1dindex++;
		total+=factor;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(d_G1D), G1D, sizeof(G1D));
}



__global__
void calcGaussianXYPlaneConvolution256(float* d_volumeChunk,float* d_tempVolumeChunk)
{
  //assume each plane is 256x256
	//  each thread does a full column
	uint plane = blockIdx.x;       //the plane
	uint col = threadIdx.x;         //the column

	__shared__ float yConvol[256];  //256

	float valCache[5];  //should be stored in reg and not lmem - but always good idea to check ptx / cubin output
	valCache[0] = 0;
	valCache[1] = 0;
	valCache[2] = 0;
	valCache[3] = 0;
	valCache[4] = 0;

	for (int c=0;c<256;++c)
	{
		float inputVal = d_volumeChunk[plane*256*256+c*256+col];  
		
		valCache[0]=valCache[1];
		valCache[1]=valCache[2];
		valCache[2]=valCache[3];
		valCache[3]=valCache[4];
		valCache[4]=inputVal;
		
		if (c>1)    //we have enough data to produce a value
		{
			float outputVal=valCache[0]*d_G1D[0]+valCache[1]*d_G1D[1]+valCache[2]*d_G1D[2]+valCache[3]*d_G1D[3]+valCache[4]*d_G1D[4];
			yConvol[col] = outputVal;

			__syncthreads(); //dangerous to put this in a loop/conditional but in this case should be fine - can probably do without it as 1/2 warp will be sync'd

			//now calc the x convol
			float xConvol = 0;
			if ((col>1) && (col<256-1))
			{
			   xConvol  = yConvol[col-2]*d_G1D[0]+yConvol[col-1]*d_G1D[1]+yConvol[col]*d_G1D[2]+yConvol[col+1]*d_G1D[3]+yConvol[col+2]*d_G1D[4];
			}

			//__syncthreads(); - not really needed here but syncs the writes
			d_tempVolumeChunk[plane*256*256+c*256+col] = xConvol;  //coalesced
		}
	}

}
